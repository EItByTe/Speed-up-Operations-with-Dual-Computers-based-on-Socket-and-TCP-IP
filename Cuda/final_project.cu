#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <math.h>
#include <Windows.h>
#include "hip/device_functions.h"
#include <omp.h>
#define DATA_SIZE 128000000
#define THREAD_NUM 256
// Block����
#define BLOCK_NUM 32
#define Row 8000
#define Col 8000       //�������Ķ�ά����ṹΪ8000*8000��Ϊ�����ݵ�һ��
#define MAX_ONCE(a,b) (((a) > (b)) ? (a):(b))
float data[DATA_SIZE];
int clockRate;

/* ��������� */
void generateNumbers(float* numbers, int size) {
    int i;
    for (i = 0; i < size; i++) {
        numbers[i] = float(rand() + rand() + rand() + rand());
    }
}

/* ��ӡGPU�豸��Ϣ */
void printDeviceProps(const hipDeviceProp_t* prop) {
    printf("Device Name: %s\n", prop->name);
    printf("totalGlobalMem: %ld\n", prop->totalGlobalMem);
    printf("sharedMemPerBlock: %d\n", prop->sharedMemPerBlock);
    printf("regsPerBlock: %d\n", prop->regsPerBlock);
    printf("warpSize: %d\n", prop->warpSize);
    printf("memPitch: %d\n", prop->memPitch);
    printf("maxThreadPerBlock: %d\n", prop->maxThreadsPerBlock);
    printf("maxThreadsDim[0-2]: %d %d %d\n", prop->maxThreadsDim[0], prop->maxThreadsDim[1], prop->maxThreadsDim[2]);
    printf("maxGridSize[0-2]: %d %d %d\n", prop->maxGridSize[0], prop->maxGridSize[1], prop->maxGridSize[2]);
    printf("totalConstMem: %d\n", prop->totalConstMem);
    printf("major: %d & minor: %d\n", prop->major, prop->minor);
    printf("clockRate: %d\n", prop->clockRate); clockRate = prop->clockRate;
    printf("textureAlignment: %d\n", prop->textureAlignment);
    printf("deviceOverlap: %d\n", prop->deviceOverlap);
    printf("multiProcessorCount: %d\n", prop->multiProcessorCount);
}

/* CUDA ��ʼ�� */
bool initCUDA() {
    int count, i;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);

    if (0 == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    for (i = 0; i < count; i++) {
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    hipSetDevice(i);

    //printDeviceProps(&prop);

    return true;
}

float pure_max(const float data[], const int len)
{
    double max_temp = 0;
    for (int i = 0; i < len; i++) {
        if (log(sqrt(data[i])) > max_temp)
            max_temp = log(sqrt(data[i]));
    }
    return float(max_temp);
}
/* ��������ʱ */
clock_t findMaxTimeUsed(const clock_t* time) {
    int i;
    clock_t min_start = time[0], max_end = time[BLOCK_NUM];
    for (i = 0; i < BLOCK_NUM; i++) {
        if (time[i] < min_start) {
            min_start = time[i];
        }
        if (time[i + BLOCK_NUM] > max_end) {
            max_end = time[i + BLOCK_NUM];
        }
    }
    return max_end - min_start;
}

/* ����ͣ�__global__����������GPU��*/
__global__ static void sumOfSquares(float* numbers, float* sub_sum, clock_t* time) {
    int i;

    // ��ȡ��ǰ�߳�������Block�ţ���0��ʼ��
    const int block_id = blockIdx.x;
    const int thread_id = threadIdx.x;

    if (thread_id == 0) {
        time[block_id] = clock();
    }

    sub_sum[block_id * THREAD_NUM + thread_id] = 0;
    // Block0-�߳�0��ȡ��0��Ԫ�أ�Block0-�߳�1��ȡ��1��Ԫ��...Block1-�߳�0��ȡ��THREAD_NUM��Ԫ�أ��Դ�����... 
    for (i = block_id * THREAD_NUM + thread_id; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
        sub_sum[block_id * THREAD_NUM + thread_id] += log10f(sqrtf(numbers[i]));
    }

    if (thread_id == 0) {
        time[block_id + BLOCK_NUM] = clock();
    }
}

/* ����ͣ�__global__����������GPU��*/
__global__ static void cal_max(float* numbers, float* sub_sum, clock_t* time) {
    int i;

    // ��ȡ��ǰ�߳�������Block�ţ���0��ʼ��
    const int block_id = blockIdx.x;
    const int thread_id = threadIdx.x;

    if (thread_id == 0) {
        time[block_id] = clock();
    }

    sub_sum[block_id * THREAD_NUM + thread_id] = 0;
    // Block0-�߳�0��ȡ��0��Ԫ�أ�Block0-�߳�1��ȡ��1��Ԫ��...Block1-�߳�0��ȡ��THREAD_NUM��Ԫ�أ��Դ�����... 
    for (i = block_id * THREAD_NUM + thread_id; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
        sub_sum[block_id * THREAD_NUM + thread_id] += log10f(sqrtf(numbers[i]));
    }

    if (thread_id == 0) {
        time[block_id + BLOCK_NUM] = clock();
    }
}

__global__ void cudacalculate(float** C, float** A)
//����cuda���ټ��㺯��cudacalculate
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    //�����̵߳��̱߳��
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < Col && idy < Row) {
        C[idy][idx] = log(sqrt(A[idy][idx]));
        //��������λ��ƽ��ȡlog��ָ���������
    }
}

__global__ void maxCuda(float d_a[DATA_SIZE], float d_a_temp[DATA_SIZE], float dat_Max[BLOCK_NUM])
{
    __shared__ float Max_temp[THREAD_NUM];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
     int tid = threadIdx.x;
    d_a_temp[i] = log(sqrt(d_a[i]));
    Max_temp[tid] = d_a_temp[tid + blockIdx.x * blockDim.x];
    __syncthreads();
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tid < stride) {
            float temp1 = Max_temp[tid + stride];
            if (Max_temp[tid] < temp1) {
                float temp = Max_temp[tid];
                Max_temp[tid] = temp1;
                Max_temp[tid + stride] = temp;
                }
        }
    }
        if (tid == 0)
            dat_Max[blockIdx.x] = Max_temp[0];
}


int main(void) {
    if (!initCUDA()) {
        return 0;
    }
    float* gpudata;
    int i;
    double sum;
    float sub_sum[BLOCK_NUM * THREAD_NUM], * gpu_sub_sum;
    // ÿ��Block����һ����ʱ��Ԫ
    clock_t time_used[BLOCK_NUM * 2], * gpu_time_used, start, finish;
    float cpucosttime;
    generateNumbers(data, DATA_SIZE);

    /*------------------------------------------------*/
    float* Mat_d;
    float* Mat_d_temp;
    float* dat_Max;
    float ret_Max[BLOCK_NUM];
    hipMalloc((void**)&Mat_d, DATA_SIZE * sizeof(float));
    hipMalloc((void**)&Mat_d_temp, DATA_SIZE * sizeof(float));
    hipMalloc((void**)&dat_Max, DATA_SIZE * sizeof(float));
    start = clock();
    hipMemcpy(Mat_d, data, DATA_SIZE * sizeof(float), hipMemcpyHostToDevice);//�����ݴ�Host ���� Device
    maxCuda << <BLOCK_NUM, THREAD_NUM >> > (Mat_d, Mat_d_temp, dat_Max); //�����ں˺���
    hipMemcpy(ret_Max, dat_Max, sizeof(float) * BLOCK_NUM, hipMemcpyDeviceToHost);//��������ص�������
    //ʹ�� cpu ���߳�ȡ���������е����ֵ
    float max = 0;
    float maxx[BLOCK_NUM] = {0};
    for (int i = 0; i < BLOCK_NUM; i++) {
        maxx[i] = MAX_ONCE(maxx[i], ret_Max[i]);
    }
    for (int i = 0; i < BLOCK_NUM; i++)
        max = MAX_ONCE(maxx[i], max);
    finish = clock();
    double gpumaxcosttime = (float)(finish - start) / CLOCKS_PER_SEC;
    printf("GPU max is: %f, time used : %lf(s)\n", max, gpumaxcosttime);
    //�ͷ��Դ�ռ�
    hipFree(Mat_d);
    hipFree(Mat_d_temp);
    hipFree(dat_Max);

    float max_cpu;
    /*--------------�޼��������ֵ��ʼ--------------*/
    start = clock();
    max_cpu = pure_max(data, DATA_SIZE);
    finish = clock();
    float cpumaxcosttime = (float)(finish - start) / CLOCKS_PER_SEC;
    printf("CPU max is: %f, time used : %f(s)\n", max_cpu, cpumaxcosttime);
    /*--------------�޼��������ֵ����--------------*/
    /*------------------------------------------------*/

    /*��Ͳ���*/
    hipMalloc((void**)&gpudata, sizeof(float) * DATA_SIZE);
    // ��ǰһ����BLOCK_NUM * THREAD_NUM���߳�
    hipMalloc((void**)&gpu_sub_sum, sizeof(float) * BLOCK_NUM * THREAD_NUM);
    hipMalloc((void**)&gpu_time_used, sizeof(clock_t) * BLOCK_NUM * 2);

    hipMemcpy(gpudata, data, sizeof(float) * DATA_SIZE, hipMemcpyHostToDevice);
    // ����Block��
    start = clock();
    sumOfSquares << < BLOCK_NUM, THREAD_NUM, 0 >> > (gpudata, gpu_sub_sum, gpu_time_used);

    hipMemcpy(time_used, gpu_time_used, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost);
    hipMemcpy(sub_sum, gpu_sub_sum, sizeof(float) * BLOCK_NUM * THREAD_NUM, hipMemcpyDeviceToHost);

    sum = 0.0f;

    for (i = 0; i < BLOCK_NUM * THREAD_NUM; i++) {
        sum += sub_sum[i];
    }
    finish = clock();
    float gpucosttime = (float)(finish - start) / CLOCKS_PER_SEC;
    printf("GPU sum is: %f, time used : %f(s)\n", sum, gpucosttime);
    hipFree(gpudata);
    hipFree(gpu_sub_sum);
    hipFree(time);

    //cpu
    start = clock();
    sum = 0.0f;
    LARGE_INTEGER  start2 = { 0 }; LARGE_INTEGER  end2 = { 0 };
    for (i = 0; i < DATA_SIZE; i++) {
        sum += log10f(sqrtf(data[i]));
    }
    finish = clock();
    cpucosttime = (float)(finish - start) / CLOCKS_PER_SEC;
    printf("CPU sum is: %f, time used : %f(s)\n", sum, cpucosttime);

    system("pause");

    return 0;
}